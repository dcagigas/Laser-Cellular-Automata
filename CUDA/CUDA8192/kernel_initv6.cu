#include "hip/hip_runtime.h"
#include "globals.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""
#include <stdio.h>
#include <time.h>

extern hiprandState* devStates;
extern void free_GPU_memory ();


// INIT RANDOM VALUES, ONE PER EACH CELL:
__global__ void setup_kernel_v6 (hiprandState * state, unsigned long SEED) 
{        
    int i, idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Init Random vector
    for (i = idx; i<SIDE*SIDE; i += stride)
	    hiprand_init( (SEED << 20) + i, 0, 0, &state[i]);

}


int Launcher_init_v6 ()
{
    hipError_t cudaStatus;

    // 0 - SETUP/INIT KERNEL 
    setup_kernel_v6 <<<num_blocks, num_threads>>> ( devStates, 1 );
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching setup_kernel_v6!\n", cudaStatus);
		goto Error;
    }
    
    return OK;

    Error:
        free_GPU_memory ();
        return (int) cudaStatus;
}


